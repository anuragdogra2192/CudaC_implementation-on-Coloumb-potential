
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include "stdlib.h"

//Device Code....

__global__ void force(double *deviceq,double *devicex,double *devicey,double *devicez,double *deviceFx,double *deviceFy,double *deviceFz,double *deviceU,int N) 
{	
	 double foxij,foyij,fozij,xij,yij,zij,rij;
	 int i = blockDim.x * blockIdx.x + threadIdx.x;
	 int j;
	 if(i < N-1)
         {       //Anurag Dogra
                 for(j=i;j<N;j++)
                     {
                          if(i!=j)
                          {
                              xij = devicex[i] - devicex[j];
                              yij = devicey[i] - devicey[j];
                              zij = devicez[i] - devicez[j];

                              //Distance calculation
                              rij = sqrt((xij*xij)+(yij*yij)+(zij*zij));

                              foxij = foxij + ((deviceq[i]*deviceq[j]*xij)/(rij*rij*rij));
                              foyij = foyij + ((deviceq[i]*deviceq[j]*yij)/(rij*rij*rij));
                              fozij = fozij + ((deviceq[i]*deviceq[j]*zij)/(rij*rij*rij));

                              deviceFx[i] = deviceFx[i] + foxij;
                              deviceFy[i] = deviceFy[i] + foyij;
                              deviceFz[i] = deviceFz[i] + fozij;
                              deviceU[i] = deviceU[i] + 2*(deviceq[j]/rij);

                           }
                    }
         }
                       
} 
int main(int argc, char** argv)
{	
	int device = 0;
	if(argc > 1)
	{	
		device = atoi(argv[1]);
	}
	
	hipSetDevice(device);
	int N;
	double *Fx,*Fy,*Fz,*U;
	int i,j,k;
	double *q,*x,*y,*z;
	//device Arrays......
	double *deviceq,*devicex,*devicey,*devicez,*deviceU;
	double *deviceFx,*deviceFy,*deviceFz;
	// INPUT WORK ..
	N = 1000;
	double Ec = 0;
	hipMallocManaged(&deviceq, N * sizeof(double));
	hipMallocManaged(&devicex, N * sizeof(double));
	hipMallocManaged(&devicey, N * sizeof(double));
	hipMallocManaged(&devicez, N * sizeof(double));
	hipMallocManaged(&deviceU, N * sizeof(double));
        hipMallocManaged(&deviceFx, N * sizeof(double));
	hipMallocManaged(&deviceFy, N * sizeof(double));
	hipMallocManaged(&deviceFz, N * sizeof(double));

	q=(double*)malloc((N)*sizeof(double));
	x=(double*)malloc((N)*sizeof(double));
	y=(double*)malloc((N)*sizeof(double));
	z=(double*)malloc((N)*sizeof(double));
	//ifstream infile;//INFILE IS THE OBJECT OF ifstream class
	FILE *infileq = fopen("q.dat","r");//OPEN A FILE IN READ MODE ONLY 
        //ERROR CHECK IF THERE IS PROBLEM IN OPENING A FILE 
	for(i=0;i<N;i++)
	{	
		fscanf(infileq,"%lf",&q[i]);
	}
        FILE *infilex = fopen("x.dat","r");//OPEN A FILE IN READ MODE ONLY 
        //ERROR CHECK IF THERE IS PROBLEM IN OPENING A FILE 
        for(i=0;i<N;i++)
        {       
               fscanf(infilex,"%lf",&x[i]);
        }
        //
        FILE *infiley = fopen("y.dat","r");//OPEN A FILE IN READ MODE ONLY 
        //ERROR CHECK IF THERE IS PROBLEM IN OPENING A FILE 
        for(i=0;i<N;i++)
        {        
               fscanf(infiley,"%lf",&y[i]);
        }
        //
        FILE *infilez = fopen("z.dat","r");//OPEN A FILE IN READ MODE ONLY 
        //ERROR CHECK IF THERE IS PROBLEM IN OPENING A FILE 
        for(i=0;i<N;i++)
        {        
            fscanf(infilez,"%lf",&z[i]);
        }
        //_________________________________________________________________________
	/*for(i=0;i<N;i++{	
		printf("%lf\t %lf\t %lf\t %lf\n",q[i],x[i],y[i],z[i]);
	       //cout<<q[i]<<" "<<x[i]<<" "<<y[i]<<" "<<z[i]<<endl;
	}*/

	//Copying from host array to device array
	hipMemcpy(deviceq, q, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devicex, x, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devicey, y, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devicez, z, N * sizeof(double), hipMemcpyHostToDevice);        
	
	Fx=(double*)malloc((N)*sizeof(double));
	Fy=(double*)malloc((N)*sizeof(double));
	Fz=(double*)malloc((N)*sizeof(double));
	U=(double*)malloc((N)*sizeof(double));

	for(i = 0;i < N;i++)
	{	
		deviceFx[i]=0;
		deviceFy[i]=0;
		deviceFz[i]=0;
		deviceU[i]=0;
		Fx[i]=0;
		Fy[i]=0;
		Fz[i]=0;
		U[i]=0;
	}
	dim3 blockDim(N/4);
	dim3 gridDim(4);
	// Calling the kernal
        force<<<gridDim,blockDim>>>(deviceq,devicex,devicey,devicez,deviceFx,deviceFy,deviceFz,deviceU,N);
	
	hipMemcpy(Fx, deviceFx, N * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(Fy, deviceFy, N * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(Fz, deviceFz, N * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(U,  deviceU, N * sizeof(double), hipMemcpyDeviceToHost);

	for(i=0;i<N;i++)
	{	
		Ec = Ec + q[i]*U[i];
	}
	Ec = Ec/2;
	printf("%lf\n",Ec);
	
	return 0;

}
	
	 
